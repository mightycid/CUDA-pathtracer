#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2014 MightyCid
 *
 * This file is part of CUDA-pathtracer <github.com/mightycid/CUDA-pathtracer>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>. 
 */

#include "pathtracer.h"
#include "cutil.h"

#include <>
#include <hip/hip_runtime.h>
#include <vector>
#include <time.h>

// gamme correction macro
#define GAMMA_FLOAT(c, gamma) powf(c, 1.f/gamma)

// saves the random numbers for subsampling for a pixel
// each thread produces one to abstract the uniform grid
struct CameraSamples {
	CUDA_DEVICE CameraSamples(float *s) : samples(s) {
		//TODO elegant way to do this
		// upper left
		samples[0] *= -0.5f;
		samples[1] *= -0.5f;
		// upper right
		samples[2] *= 0.5f;
		samples[3] *= -0.5f;
		// lower left
		samples[4] *= -0.5f;
		samples[5] *= 0.5f;
		// lower right
		samples[6] *= 0.5f;
		samples[7] *= 0.5f;
	}
	CUDA_DEVICE UVSample operator[](uint32_t index) const {
		return UVSample(samples[index*2], samples[index*2+1]);
	}

	float *samples;
};

__device__ Color Trace(const Ray &ray, const Scene *scene, float* rng,
	int maxBounces=10);

__global__ void GenerateRayPool(Camera camera, Ray *rayBuffer, float *devRand) {
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t index = (y*camera.Width()+x);

	if(x >= camera.Width() || y >= camera.Height()) {
		return;
	}
	
	CameraSamples samples(&devRand[index*4*2]);
	for(int i=0; i<4; ++i)
		rayBuffer[index*4+i] = camera.GenerateRay(x, y, samples[i]);
}

__global__ void RenderKernel(const Scene *scene, float *buffer, 
		float *rng, Ray *rayPool, uint32_t width, uint32_t height, 
		uint32_t iteration, int maxBounces) {
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t index = (y*width+x);
	const uint32_t fbindex = index*3;

	// leave thread if out of bounds
	if(x >= width || y >= height)
		return;

	int rngOffset = index*maxBounces*SAMPLES_PER_BOUNCE;

	Color c;
	// 2x2 Subsampling
	for(int i=0; i<4; ++i) {
		Ray ray = rayPool[index*4+i];
		c += Trace(ray, scene, &rng[rngOffset], maxBounces);
	}
	c /= 4.f;

	float gamma = 1.0f;
	float invIteration = 1.f/(float)(iteration);
	
	buffer[fbindex]   = (buffer[fbindex]   * (float)(iteration-1)
		+ c.r) * invIteration;
	buffer[fbindex+1] = (buffer[fbindex+1] * (float)(iteration-1)
		+ c.g) * invIteration;
	buffer[fbindex+2] = (buffer[fbindex+2] * (float)(iteration-1)
		+ c.b) * invIteration;
}

__device__ Color Trace(const Ray &r, const Scene *scene, float* rng, 
		int maxBounces) {
	Color pathThroughput(1.f,1.f,1.f), L;

	Ray ray = r;
	bool specularBounce = false;
	Intersection isect;

	if(!Intersect(scene, ray, &isect))
		return Color();

	Intersection isectp = isect;

	for (int bounces = 0;; ++bounces) {
		const Primitive *prim = isectp.prim;
		const Material *mat = scene->materials[prim->materialId];
		const Point &p = isectp.p;
		const Vec &n = isectp.n;
		const Vec wo = ray.d;
		const int rngIndex = bounces*SAMPLES_PER_BOUNCE;

		//if ((bounces == 0 || specularBounce) && prim->IsLight()) {
		if (prim->IsLight()) {
			Light *light = scene->lights[prim->lightId];
			L += pathThroughput * light->L(p, wo, n);
		}
		
		Vec wi;
		float pdf;
		UVSample bsdfSample = UVSample(rng[rngIndex], rng[rngIndex+1]);
		// get reflected sample
		const Color f = SampleMaterial(mat, wo, &wi, &pdf, isectp.n, bsdfSample);

		// leave if current sample gives no contribution
		if (f.IsBlack() || pdf == 0.f) break;

		specularBounce = (mat->type == SPECULAR || mat->type == TRANSMISSIVE);
		pathThroughput *= f * fabs(wi.Dot(n)) / pdf;
		ray = Ray(p, wi);

		// russian roulette
		if (bounces > 3) {
			float continueProbability = min(.5f, pathThroughput.Max());
			float rnd = rng[rngIndex+2];
			if (rnd > continueProbability)
				break;
			pathThroughput /= continueProbability;
		}
		if (bounces == maxBounces)
			break;

		// if ray leaves scene we can stop here
		if (!Intersect(scene, ray, &isect))
			break;

		isectp = isect;
	}
	return L;
}

bool Pathtracer::Init(const std::vector<Material> &mv, 
		const std::vector<Primitive> &pv, const std::vector<Light> &lv) {

	//copy primitive list to device
	Primitive* devPrimitives = NULL;
	size_t numPrims = pv.size();
	CudaSafeCall(hipMalloc((void**)&(devPrimitives), 
		numPrims*sizeof(Primitive)));
	CudaSafeCall(hipMemcpy(devPrimitives, &pv[0], numPrims*sizeof(Primitive),
		hipMemcpyHostToDevice));
	PrimitiveList primList = PrimitiveList(devPrimitives, numPrims);

	//copy light list to device
	Light* devLights = NULL;
	size_t numLights = lv.size();
	CudaSafeCall(hipMalloc((void**)&(devLights), numLights*sizeof(Light)));
	CudaSafeCall(hipMemcpy(devLights, &lv[0], numLights*sizeof(Light), 
		hipMemcpyHostToDevice));
	LightList lightList = LightList(devLights, numLights);

	//copy material list to device
	Material* devMats = NULL;
	size_t numMats = pv.size();
	CudaSafeCall(hipMalloc((void**)&(devMats), numMats*sizeof(Material)));
	CudaSafeCall(hipMemcpy(devMats, &mv[0], numMats*sizeof(Material), 
		hipMemcpyHostToDevice));
	MaterialList matList = MaterialList(devMats, numMats);

	//copy scene to device
	Scene hostScene (matList, primList, lightList);
	CudaSafeCall(hipMalloc((void**)&(scene), sizeof(Scene)));
	CudaSafeCall(hipMemcpy(scene, &hostScene, sizeof(Scene), 
		hipMemcpyHostToDevice));

	//allocate memory for random numbers
	sampleSize = width*height*SAMPLES_PER_PIXEL*maxBounces*SAMPLES_PER_BOUNCE;
	CudaSafeCall(hipMalloc((void**)&(devRand), sampleSize*sizeof(float)));

	//allocate memory for ray pool
	rayPool = NULL;
	size_t numRays = width*height*4;
	CudaSafeCall(hipMalloc((void**)&(rayPool), numRays*sizeof(Ray)));

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
	CudaCheckError();

	return true;
}

void Pathtracer::Run(float* devBuffer) {
	// create random numbers
	hiprandGenerateUniform(gen, devRand, sampleSize);
	CudaCheckError();

	int dimx = 16;
	int dimy = 16;
	dim3 dimGrid(width/dimx, height/dimy);
	dim3 dimBlock(dimx, dimy);

	GenerateRayPool<<<dimGrid, dimBlock>>>(*camera, rayPool, devRand);
	hipDeviceSynchronize();
	CudaCheckError();

	//launch kernel
	uint32_t randOffset = width*height*SAMPLES_PER_PIXEL;
	RenderKernel<<<dimGrid, dimBlock>>>(scene, devBuffer, &devRand[randOffset],
		rayPool, width, height, ++iteration, maxBounces);

	hipDeviceSynchronize();
	CudaCheckError();
}

void Pathtracer::Reset() {
	iteration = 0;
}

void Pathtracer::Release() {
	//release buffers
	hiprandDestroyGenerator(gen);
	hipFree(devRand);
	hipFree(rayPool);
	hipFree(scene->lights.lights);
	hipFree(scene->materials.materials);
	hipFree(scene->primitives.primitives);
	hipFree(scene);
}
