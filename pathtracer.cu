#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2014 MightyCid
 *
 * This file is part of CUDA-pathtracer <github.com/mightycid/CUDA-pathtracer>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>. 
 */

#include "scene.h"
#include "pathtracer.h"
#include "cutil.h"

#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <vector>
#include <time.h>

// gamme correction macro
#define GAMMA_FLOAT(c, gamma) powf(Clamp(c, 0.f, 1.f), 1.f/gamma)

// saves the random numbers for subsampling for a pixel
// each thread produces one to abstract the uniform grid
struct CameraSamples {
	CUDA_DEVICE CameraSamples(float *s) : samples(s) {
		//TODO elegant way to do this
		// upper left
		samples[0] *= -0.25f;
		samples[1] *= -0.25f;
		// upper right
		samples[2] *= 0.25f;
		samples[3] *= -0.25f;
		// lower left
		samples[4] *= -0.25f;
		samples[5] *= 0.25f;
		// lower right
		samples[6] *= 0.25f;
		samples[7] *= 0.25f;
	}
	CUDA_DEVICE CameraSample operator[](uint32_t index) const {
		return CameraSample(samples[index*2], samples[index*2+1]);
	}

	float *samples;
};


__device__ Color Trace(const Ray &ray, const Scene *scene, float* rng, int maxBounces=10);

__global__ void kernel(const Scene *scene, float *buffer, uint32_t bufferSize, float *rng, uint32_t iteration, int maxBounces) {
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t index = (y*scene->camera.Width()+x);
	const uint32_t fbindex = index*3;

	if(fbindex >= bufferSize)
		return;

	int rngOffset = index*4*maxBounces*3;
	int numCameraSamples = 4*2;
	CameraSamples samples(&rng[rngOffset]);

	Color c;
	// 2x2 Subsampling
	for(int i=0; i<4; ++i) {
		Ray ray = scene->camera.GenerateRay(x, y, samples[i]);
		c += Trace(ray, scene, &rng[rngOffset+numCameraSamples], maxBounces);
	}
	c /= 4.f;

	float gamma = 2.2f;
	float invIteration = 1.f/(float)iteration;
	buffer[fbindex]   = ((buffer[fbindex]   * (iteration-1) + GAMMA_FLOAT(c.r, gamma)) * invIteration);
	buffer[fbindex+1] = ((buffer[fbindex+1] * (iteration-1) + GAMMA_FLOAT(c.g, gamma)) * invIteration);
	buffer[fbindex+2] = ((buffer[fbindex+2] * (iteration-1) + GAMMA_FLOAT(c.b, gamma)) * invIteration);
}

__device__ Color Trace(const Ray &r, const Scene *scene, float* rng, int maxBounces) {
	Color pathThroughput(1.f,1.f,1.f), L;

	Ray ray = r;
	bool specularBounce = false;
	Intersection isect;

	if(!scene->Intersect(ray, &isect))
		return Color();

	Intersection isectp = isect;

	for (int bounces = 0;; ++bounces) {
		const Primitive *prim = isectp.prim;
		const Material &mat = scene->materials[prim->GetMaterialId()];
		const Point &p = isectp.p;
		const Vec &n = isectp.n;
		const Vec wo = ray.d;
		const int rngIndex = bounces*3;

		//if (bounces == 0 || specularBounce)
		//	L += pathThroughput * mat.color * (mat.emitting ? 1.f : 0.f);

		float lightPdf = 1.f;
		// chose light depend on power heuristic
		//uint32_t lightNum = lightDistribution->SampleDiscrete(rndFloat(),
		//	&lightPdf);
		//const Light *light = scene->lights[lightNum];
		const Light &light = scene->lights[0];

		BSDFSample bsdfSample(rng[rngIndex], rng[rngIndex+1]);

		// get direct lighting
		L += pathThroughput * scene->EstimateDirect(light, wo, isectp) / lightPdf;

		Vec wi;
		float pdf;
		// get reflected sampling
		const Color f = mat.SampleF(wo, &wi, &pdf, isectp, bsdfSample);

		// leave if current sample gives no contribution
		if (f.IsBlack() || pdf == 0.f) break;

		specularBounce = (mat.type == SPECULAR || mat.type == TRANSMISSIVE);
		pathThroughput *= f * fabs(wi.Dot(n)) / pdf;
		ray = Ray(p, wi);

		// russian roulette
		if (bounces > 3) {
			float continueProbability = min(.5f, pathThroughput.Y());
			float rnd = rng[rngIndex+2];
			if (rnd > continueProbability)
				break;
			pathThroughput /= continueProbability;
		}
		if (bounces == maxBounces)
			break;

		// if ray leaves scene we can stop here
		if (!scene->Intersect(ray, &isect))
			break;

		isectp = isect;
	}
	return L;
}

bool Pathtracer::Init(const Camera &camera, const std::vector<Material> &mv, const std::vector<Primitive> &pv, const std::vector<Light> &lv) {

	//copy primitive list to device
	Primitive* devPrimitives = NULL;
	size_t numPrims = pv.size();
	CudaSafeCall(hipMalloc((void**)&(devPrimitives), numPrims*sizeof(Primitive)));
	CudaSafeCall(hipMemcpy(devPrimitives, &pv[0], numPrims*sizeof(Primitive), hipMemcpyHostToDevice));
	PrimitiveList primList = PrimitiveList(devPrimitives, numPrims);

	//copy light list to device
	Light* devLights = NULL;
	size_t numLights = lv.size();
	CudaSafeCall(hipMalloc((void**)&(devLights), numLights*sizeof(Light)));
	CudaSafeCall(hipMemcpy(devLights, &lv[0], numLights*sizeof(Light), hipMemcpyHostToDevice));
	LightList lightList = LightList(devLights, numLights);

	//copy material list to device
	Material* devMats = NULL;
	size_t numMats = pv.size();
	CudaSafeCall(hipMalloc((void**)&(devMats), numMats*sizeof(Material)));
	CudaSafeCall(hipMemcpy(devMats, &mv[0], numMats*sizeof(Material), hipMemcpyHostToDevice));
	MaterialList matList = MaterialList(devMats, numMats);

	//copy scene to device
	Scene hostScene (camera, matList, primList, lightList);
	CudaSafeCall(hipMalloc((void**)&(scene), sizeof(Scene)));
	CudaSafeCall(hipMemcpy(scene, &hostScene, sizeof(Scene), hipMemcpyHostToDevice));

	//allocate memory for random numbers
	sampleSize = width*height*4*2*maxBounces*3;
	CudaSafeCall(hipMalloc((void**)&(devRand), sampleSize*sizeof(float)));

	return true;
}

void Pathtracer::Run(float* devBuffer) {
	// create random numbers
	uint32_t sampleSize = width*height*4*2*maxBounces*3;
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)); //samples=seed
	hiprandGenerateUniform(gen, devRand, sampleSize);
	hiprandDestroyGenerator(gen);
	CudaCheckError();

	int dimx = 16;
	int dimy = 16;
	dim3 dimGrid(width/dimx, height/dimy);
	dim3 dimBlock(dimx, dimy);

	//launch kernel
	kernel<<<dimGrid, dimBlock>>>(scene, devBuffer, bufferSize, devRand, ++iteration, maxBounces);
	hipDeviceSynchronize();
	CudaCheckError();
}

void Pathtracer::Release() {
	//release buffers
	hipFree(devRand);
}
